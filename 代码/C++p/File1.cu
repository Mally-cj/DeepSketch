#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>

//1M
#define DATA_SIZE 10489

#define THREAD_NUM 256

#define BLOCK_NUM 32

int data[DATA_SIZE];

//产生大量0-9之间的随机数
void GenerateNumbers(int* number, int size)
{
    for (int i = 0; i < size; i++) {
        number[i] = rand() % 10;
    }
}

//打印设备信息
void printDeviceProp(const hipDeviceProp_t& prop)
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %d.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %d.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %d.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

//CUDA 初始化
bool InitCUDA()
{
    int count;

    //取得支持Cuda的装置的数目
    hipGetDeviceCount(&count);

    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;

    for (i = 0; i < count; i++) {

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        //打印设备信息
        printDeviceProp(prop);

        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }

    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}


// __global__ 函数 (GPU上执行) 计算立方和
__global__ static void sumOfSquares(int* num, int* result)
{
    //声明一块共享内存
    extern __shared__ int shared[];

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    shared[tid] = 0;
    //thread需要同时通过tid和bid来确定，同时不要忘记保证内存连续性
    for (int i = bid * THREAD_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM) {
        shared[tid] += num[i] * num[i] * num[i];
    }

    //同步 保证每个 thread 都已经把结果写到 shared[tid] 里面
    __syncthreads();

    //使用线程0完成加和
    if (tid == 0)
    {
        for (int i = 1; i < THREAD_NUM; i++)
        {
            shared[0] += shared[i];
        }
        result[bid] = shared[0];
    }
}

// __global__ 函数 (GPU上执行) 计算立方和
__global__ static void sumOfSquares1(int* num, int* result)
{
    /*
    使用树状加法和并行计算和
    */
    extern __shared__ int shared[];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    shared[tid] = 0;

    for (int i = bid * THREAD_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM) {
        shared[tid] += num[i] * num[i] * num[i];
    }
    __syncthreads();

    //树状加法
    int offset = 1, mask = 1;
    while (offset < THREAD_NUM)
    {
        if ((tid & mask) == 0)
        {
            shared[tid] += shared[tid + offset];
        }
        offset += offset;
        mask = offset + mask;
        __syncthreads();
    }

    //计算时间,记录结果，只在 thread 0（即 threadIdx.x = 0 的时候）进行，每个 block 都会记录开始时间及结束时间
    if (tid == 0)
    {
        result[bid] = shared[0];
    }
}



int main()
{
    //生成随机数
    GenerateNumbers(data, DATA_SIZE);

    /*把数据复制到显卡内存中*/
    int* gpudata, * result;

    clock_t* time;

    //hipMalloc 取得一块显卡内存 ( 其中result用来存储计算结果，time用来存储运行时间 )
    hipMalloc((void**)&gpudata, sizeof(int) * DATA_SIZE);
    hipMalloc((void**)&result, sizeof(int) * BLOCK_NUM);

    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

    // 在CUDA 中执行函数 语法：函数名称<<<block 数目, thread 数目, shared memory 大小>>>(参数...);
    sumOfSquares1 << < BLOCK_NUM, THREAD_NUM, THREAD_NUM * sizeof(int) >> > (gpudata, result);

    int *sum = (int*)malloc(sizeof(int) * BLOCK_NUM);

    //hipMemcpy 将结果从显存中复制回内存
    hipMemcpy(&sum, result, sizeof(int) * BLOCK_NUM, hipMemcpyDeviceToHost);

    //Free
    hipFree(gpudata);
    hipFree(result);

    int final_sum = 0;
    printf("len %d\n", sizeof(sum));
    for (int i = 0; i < BLOCK_NUM; i++) {

        //final_sum += sum[i];

    }
    free(sum);
    //sum = NULL;

    printf("GPUsum: %d \n", final_sum);

    final_sum = 0;

    for (int i = 0; i < DATA_SIZE; i++) {

        final_sum += data[i] * data[i] * data[i];

    }

    printf("CPUsum: %d \n", final_sum);

    return 0;
}